
#include <hip/hip_runtime.h>
#include<cstdlib>
#include<stdio.h>
using namespace std;

__device__ void getGausspoints(double *gausspoints,double *weights,int nint)
{
//cout<<"nint "<<nint<<endl;
switch (nint)
{
	case 1:
	{
	 double xk[1]={0.000000000000000000000000};
	 double wk[1]={2.000000000000000000000000};
	gausspoints[0]=xk[1];
	weights[0]=wk[0];
	
	}
	break;
	case 2:
	{ double x[1] = {0.5773502691896257645091488};
	 double w[1] = {1.0000000000000000000000000};
	for(int i=0;i<nint/2;i++)
	{
		gausspoints[2*i]=x[i];
		gausspoints[2*i+1]=-x[i];
		weights[2*i]=w[i];
		weights[2*i+1]=w[i];
	}}
	break;
	case 4:
	{ double x[2] = {0.3399810435848562648026658,0.8611363115940525752239465};
	 double w[2] = {0.6521451548625461426269361,0.3478548451374538573730639};
	for(int i=0;i<nint/2;i++)
	{
		gausspoints[2*i]=x[i];
		gausspoints[2*i+1]=-x[i];
		weights[2*i]=w[i];
		weights[2*i+1]=w[i];
	}
	}
	break;
	case 6:
	{ double x[3] = {0.2386191860831969086305017,0.6612093864662645136613996,0.9324695142031520278123016};
	 double w[3] = {0.4679139345726910473898703,0.3607615730481386075698335,0.1713244923791703450402961};
	for(int i=0;i<nint/2;i++)
	{
		gausspoints[2*i]=x[i];
		gausspoints[2*i+1]=-x[i];
		weights[2*i]=w[i];
		weights[2*i+1]=w[i];
	}
	}
	break;
	case 8:
	{ double x[4] = {0.1834346424956498049394761,0.5255324099163289858177390,0.7966664774136267395915539,0.9602898564975362316835609};
	 double w[4] = {0.3626837833783619829651504,0.3137066458778872873379622,0.2223810344533744705443560,0.1012285362903762591525314};
		for(int i=0;i<nint/2;i++)
	{
		gausspoints[2*i]=x[i];
		gausspoints[2*i+1]=-x[i];
		weights[2*i]=w[i];
		weights[2*i+1]=w[i];
	}
	}
	break;
	case 10:
	{ double x[5] = {0.1488743389816312108848260,0.4333953941292471907992659,0.6794095682990244062343274,0.8650633666889845107320967,0.9739065285171717200779640};
	 double w[5] = {0.2955242247147528701738930,0.2692667193099963550912269,0.2190863625159820439955349,0.1494513491505805931457763,0.0666713443086881375935688};
		for(int i=0;i<nint/2;i++)
	{
		gausspoints[2*i]=x[i];
		gausspoints[2*i+1]=-x[i];
		weights[2*i]=w[i];
		weights[2*i+1]=w[i];
	}
	}
	break;
	case 12:
	{ double x[6] = {0.1252334085114689154724414,0.3678314989981801937526915,0.5873179542866174472967024,0.7699026741943046870368938,0.9041172563704748566784659,0.9815606342467192506905491};
	 double w[6] = {0.2491470458134027850005624,0.2334925365383548087608499,0.2031674267230659217490645,0.1600783285433462263346525,0.1069393259953184309602547,0.0471753363865118271946160};
		for(int i=0;i<nint/2;i++)
	{
		gausspoints[2*i]=x[i];
		gausspoints[2*i+1]=-x[i];
		weights[2*i]=w[i];
		weights[2*i+1]=w[i];
	}
	}
	break;
	case 3:
	{ double x[2] = {0.0000000000000000000000000,0.7745966692414833770358531};
	 double w[2] = {0.8888888888888888888888889,0.5555555555555555555555556};
	gausspoints[0]=x[0];weights[0]=w[0];
	for(int i=0;i<(nint-1)/2;i++)
	{
	gausspoints[2*i+1]=x[i+1];
	gausspoints[2*i+2]=-x[i+1];
	weights[2*i+1]=w[i+1];
	weights[2*i+2]=w[i+1];
	}
	}
	break;
	case 5:
	{ double x[3] = {0.0000000000000000000000000,0.5384693101056830910363144,0.9061798459386639927976269};
	 double w[3] = {0.5688888888888888888888889,0.4786286704993664680412915,0.2369268850561890875142640};
	gausspoints[0]=x[0];weights[0]=w[0];
	for(int i=0;i<(nint-1)/2;i++)
	{
	gausspoints[2*i+1]=x[i+1];
	gausspoints[2*i+2]=-x[i+1];
	weights[2*i+1]=w[i+1];
	weights[2*i+2]=w[i+1];
	}
	}
	break;
	case 7:
	{ double x[4] = {0.0000000000000000000000000,0.4058451513773971669066064,0.7415311855993944398638648,0.9491079123427585245261897};
	 double w[4] = {0.4179591836734693877551020,0.3818300505051189449503698,0.2797053914892766679014678,0.1294849661688696932706114};
	gausspoints[0]=x[0];weights[0]=w[0];
	for(int i=0;i<(nint-1)/2;i++)
	{
	gausspoints[2*i+1]=x[i+1];
	gausspoints[2*i+2]=-x[i+1];
	weights[2*i+1]=w[i+1];
	weights[2*i+2]=w[i+1];
	}
	}
	break;
	case 9:
	{ double x[5] = {0.0000000000000000000000000,0.3242534234038089290385380,0.6133714327005903973087020,0.8360311073266357942994298,0.9681602395076260898355762};
	 double w[5] = {0.3302393550012597631645251,0.3123470770400028400686304,0.2606106964029354623187429,0.1806481606948574040584720,0.0812743883615744119718922};
gausspoints[0]=x[0];weights[0]=w[0];
	for(int i=0;i<(nint-1)/2;i++)
	{
	gausspoints[2*i+1]=x[i+1];
	gausspoints[2*i+2]=-x[i+1];
	weights[2*i+1]=w[i+1];
	weights[2*i+2]=w[i+1];
	}
	}
	break;
	case 11:
	{ double x[6] = {0.0000000000000000000000000,0.2695431559523449723315320,0.5190961292068118159257257,0.7301520055740493240934163,0.8870625997680952990751578,0.9782286581460569928039380};
	 double w[6] = {0.2729250867779006307144835,0.2628045445102466621806889,0.2331937645919904799185237,0.1862902109277342514260976,0.1255803694649046246346943,0.0556685671161736664827537};
	gausspoints[0]=x[0];weights[0]=w[0];
	for(int i=0;i<(nint-1)/2;i++)
	{
	gausspoints[2*i+1]=x[i+1];
	gausspoints[2*i+2]=-x[i+1];
	weights[2*i+1]=w[i+1];
	weights[2*i+2]=w[i+1];
	}
	}
	break;
	case 13:
	{ double x[7] = {0.0000000000000000000000000,0.2304583159551347940655281,0.4484927510364468528779129,0.6423493394403402206439846,0.8015780907333099127942065,0.9175983992229779652065478,0.9841830547185881494728294};
	 double w[7] = {0.2325515532308739101945895,0.2262831802628972384120902,0.2078160475368885023125232,0.1781459807619457382800467,0.1388735102197872384636018,0.0921214998377284479144218,0.0404840047653158795200216};
	gausspoints[0]=x[0];weights[0]=w[0];
	for(int i=0;i<(nint-1)/2;i++)
	{
	gausspoints[2*i+1]=x[i+1];
	gausspoints[2*i+2]=-x[i+1];
	weights[2*i+1]=w[i+1];
	weights[2*i+2]=w[i+1];
	}
	}
	break;
	default:
		{printf("NINT not found\n");	
	//	exit(0);
		asm("trap;");
		}
}







}

