#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include"header.h"
__global__ void boundary_condtn(int *d_boundary_nodes,int maxnodes,int n_node,float end_load,int noof_endloaded_nodes,float *d_global_vector)
{
        int threadid=blockIdx.x*blockDim.x+threadIdx.x;
        if(threadid<maxnodes)
        {
        int k1=dg_global_matrix_ptr[d_boundary_nodes[threadid]];
        int k2=dg_global_matrix_ptr[d_boundary_nodes[threadid]+1];

                for(int i=0;i<(k2-k1);i++)
                {
                if(dg_global_matrix_col[k1+i]==d_boundary_nodes[threadid])
                dg_global_matrix_data[k1+i]=1;
                else
                dg_global_matrix_data[k1+i]=0;

                }

	}

        if(threadid < noof_endloaded_nodes)
        {
                end_load=end_load/noof_endloaded_nodes;
                 int k1=d_boundary_nodes[maxnodes+threadid];
                d_global_vector[k1]+=-end_load;
        }
//__syncthreads();
}
