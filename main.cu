#include "hip/hip_runtime.h"
#include<iostream>
#include<sstream> //for stringstream 
#include<cstdlib> //for exit
#include"header.h"
#include<fstream>
#include"header_cpp.h"
#include"clock.h"
#include<cusp/print.h>
#include<cusp/csr_matrix.h>
#include<cusp/krylov/cg.h>
#include<cusp/copy.h>
#include<cmath>
#include<cusp/monitor.h>
using namespace std;
//#define nint 2
	//	int *boundary_nodes;
__device__ __constant__ int *dg_connect_matrix;
__device__ __constant__ float *dg_nodal_data;
__device__ __constant__ float *dg_global_matrix_data;
__device__ __constant__ int *dg_global_matrix_col;
__device__ __constant__ int *dg_global_matrix_ptr;
__device__ __constant__ int *dg_csr_index_data;
void print(int,int,int);
template<typename T>void print(T *,int size,int rowlength);
void print(float **,int);
void Calc_global_rowptr();
void Cal_csrindex(int elementno,int *csr_index);
//float GetGPUmeminfo();
void FE(int x_nelement, int y_nelement, int z_nelement, int loop, double * rho, double penal)
{
	//hipSetDevice(0);
	StopWatch_GPU clock;
	StopWatch_CPU watch;
	
	int *boundary_nodes;
	int *re_connect_matrix;
	float *re_nodal_data;
	

	float end_load;
	float y_modulus;
	float p_ratio;

	end_load=51;
	y_modulus=1;
	//p_ratio=(float)1/3;	
	p_ratio=0.3;


	//nnz=(x_nelement*3+1)*(y_nelement*3+1)*18*2+(x_nelement*3+1)*(y_nelement*3+1)*27*(z_nelement-1);//No of nonzeros in global assembled matrix
	//cout<<endl<<nnz<<endl;
	int nelement=x_nelement*y_nelement*z_nelement;
	n_node=3*(x_nelement+1)*(y_nelement+1)*(z_nelement+1);//total no of node for 3 DOF per node
        nnz=(x_nelement*3+1)*(y_nelement*3+1)*18*2+(x_nelement*3+1)*(y_nelement*3+1)*27*(z_nelement-1);//No of nonzeros in global assembled matrix for structured mess
	cout<<"Elements "<<x_nelement<<"\t"<<y_nelement<<"\t"<<z_nelement<<endl;
////////////////////////**************************************material data initialising
//cout<<"taking isotropic material"<<"\n";

	material_data=new float[36];
	for(int i=0;i<36;i++)
	material_data[i]=0;
	{
	float lambda=p_ratio*y_modulus/((1+p_ratio)*(1-2*p_ratio));
	float shear_m=y_modulus/(2*(1+p_ratio));
	material_data[0]=material_data[7]=material_data[14]=lambda+2*shear_m;
	material_data[1]=material_data[2]=material_data[6]=material_data[8]=material_data[12]=material_data[13]=lambda;
	material_data[21]=material_data[28]=material_data[35]=shear_m;
	}
	//material_data[0]=material_data[1]=material_data[3]=material_data[4]=material_data[8]=1;
	//print(material_data,36,6);
/////************************************************************************
	int *csr_index_data=	new int[64*nelement];
	int *re_csr_index_data= new int[64*nelement];

	re_connect_matrix=	new int[x_nelement*y_nelement*z_nelement*8];
	boundary_nodes=		new int[(z_nelement+1)*(y_nelement+1)*3+y_nelement+1];//keeps fixed nodes of cantilever as well as end nodes for loading
	re_nodal_data=		new float[24*nelement];

	for(int i=0;i<576;i++)  elemental_data[i]=0;
//********************************************************************************mesh generation
	cout<<"after initialisation"<<endl;
	if(loop==1)
	{	
	data_read(n_node,nelement);
	element_stiffness1(1,elemental_data,material_data);
	}
	//print(elemental_data,576,24);

	cout<<"In middle"<<endl;		
	//meshing(d_length,d_breadth,d_height, x_nelement,y_nelement,z_nelement);
	//color_mesh(x_nelement*y_nelement*z_nelement);
	//print(x_nelement,y_nelement,z_nelement);
	int *element_color_data;//keeps the colourwise element no list
	
		int colors_size[8]={0,0,0,0,0,0,0,0};//Since at max 8 colors are used for structured mesh
	
		element_color_data=new int[x_nelement*y_nelement*z_nelement];	
		for(int i=0;i<x_nelement*y_nelement*z_nelement;i++)
		colors_size[element_color[i]]++;
		int a[8]={0};
		int k=0;
		for(int i=0;i<8;i++) a[i]=colors_size[i];
		for(int i=0;i<x_nelement*y_nelement*z_nelement;i++)
		{
			k=element_color[i];
			element_color_data[k*colors_size[k]+colors_size[k]-a[k]]=i;
			a[k]-=1;	

		}	




	for(int i=0;i<8;i++)
	cout<<colors_size[i]<<endl;

	

//for(int i=0;i<x_nelement*y_nelement*z_nelement;i++)
//cout<<element_color_data[i]<<endl;
Calc_global_rowptr();//calculate global_matrix_ptr 
Calc_boundary_nodes(boundary_nodes,x_nelement,y_nelement,z_nelement);
	for(int i=1;i<=nelement;i++)
	{
		//cout<<"yes"<<endl;
		Cal_csrindex(i,csr_index_data+(64*(i-1)));		//calculate csr_matrix data index for global assembly
		//csr_index_data+=64;	
	}
//cout<<csr_index_data[0]<<endl;
//print(csr_index_data,64*nelement,8);

watch.start();
Reorder_connectivity(element_color_data,colors_size,8,re_connect_matrix);

Reorder_nodal_data(re_connect_matrix,colors_size,8,re_nodal_data);
Reorder_csr_index(element_color_data,csr_index_data,re_csr_index_data,8,colors_size);
cout<<"reordered connectivity time :"<<watch.elapsed()<<endl;

//for(int i=0;i<576;i++) elemental_data[i]=0;
//for(int i=0; i<24*nelement;i++) nodal_data[i]=re_nodal_data[i];
//element_stiffness2(0,material_data,elemental_data,0,32768,re_nodal_data);
//print(elemental_data,576,24);




//print(csr_index_data,4*64,8);
//cout<<"Reordered index"<<endl;
//print(re_csr_index_data,4*64,8);
//print(re_nodal_data,96,1);
//print(re_connect_matrix,128,1);
//print(connect_matrix,80,1);
global_matrix_col=new int[nnz];
global_matrix_data=new float[nnz];
//print(boundary_nodes,(z_nelement+1)*(y_nelement+1)*3+y_nelement+1,1);
//cout<<boundary_nodes[0]<<endl;
//*********************************************************************************Device variables
	int *d_element_color_data;
	float *d_nodal_data;
	int *d_connect_matrix; 
	float *d_material_data;
	float *d_global_vector;
	float *d_global_matrix_data;	//Array for CSR matrix format
	float *d_sol;
	int *d_global_matrix_col;
	int *d_global_matrix_ptr;
	int *d_csr_index_data;
	int *d_boundary_nodes;
	float *d_element_matrix;
	double *d_rho;
	//int *d_re_connect_matrix;
		hipMalloc(&d_rho,              sizeof(double)*nelement);
		hipMalloc(&d_element_matrix,   sizeof(float)*576);
		hipMalloc(&d_sol,		sizeof(float)*n_node);
		hipMalloc(&d_element_color_data,sizeof(int)*nelement);
		hipMalloc(&d_nodal_data,	sizeof(float)*24*nelement);
		hipMalloc(&d_connect_matrix,	sizeof(int)*nelement*8);
		hipMalloc(&d_material_data,	sizeof(float)*36);
		hipMalloc(&d_global_matrix_data,sizeof(float)*nnz);		
		hipMalloc(&d_global_matrix_col,sizeof(int)*nnz);
		hipMalloc(&d_global_matrix_ptr,sizeof(int)*(n_node+1));
		hipMalloc(&d_csr_index_data,	sizeof(int)*64*nelement);
		hipMalloc(&d_global_vector,	sizeof(float)*n_node);
		hipMalloc(&d_boundary_nodes,	sizeof(int )*((z_nelement+1)*(y_nelement+1)*3+y_nelement+1));
	//	hipMalloc(&d_re_connect_matrix,sizeof(int)*nelement*8);	
			hipMemset(d_global_matrix_data,0,sizeof(float)*nnz);
			hipMemset(d_global_matrix_col,0,sizeof(int)*nnz);
			hipMemset(d_global_vector,0,	sizeof(float)*n_node);
			hipMemset(d_sol,0,sizeof(float)*n_node);
clock.start();
		//hipMemcpy(d_re_connect_matrix,re_connect_matrix,sizeof(int)*nelement*8,hipMemcpyHostToDevice);
		hipMemcpy(d_rho, rho,  sizeof(double)*nelement,hipMemcpyHostToDevice);
		hipMemcpy(d_element_matrix, elemental_data, sizeof(float)*576, hipMemcpyHostToDevice);
		hipMemcpy(d_element_color_data,element_color_data,sizeof(int)*nelement,hipMemcpyHostToDevice);
		hipMemcpy(d_nodal_data,re_nodal_data,sizeof(float)*nelement*24,hipMemcpyHostToDevice);
		hipMemcpy(d_connect_matrix,re_connect_matrix,sizeof(int)*nelement*8,hipMemcpyHostToDevice);
		hipMemcpy(d_material_data,material_data,sizeof(float)*36,hipMemcpyHostToDevice);
		hipMemcpy(d_csr_index_data,re_csr_index_data,sizeof(int)*64*nelement,hipMemcpyHostToDevice);
		hipMemcpy(d_global_matrix_ptr,global_rowptr,sizeof(int)*(n_node+1),hipMemcpyHostToDevice);
		hipMemcpy(d_boundary_nodes,boundary_nodes,sizeof(int)*((z_nelement+1)*(y_nelement+1)*3+y_nelement+1),hipMemcpyHostToDevice);
				hipError_t result=hipMemcpyToSymbol(HIP_SYMBOL(dg_nodal_data),&d_nodal_data,sizeof(float*));
			   hipMemcpyToSymbol(HIP_SYMBOL(dg_connect_matrix),&d_connect_matrix,sizeof(int *));
			   hipMemcpyToSymbol(HIP_SYMBOL(dg_global_matrix_data),&d_global_matrix_data,sizeof(float *));
			   hipMemcpyToSymbol(HIP_SYMBOL(dg_global_matrix_col),&d_global_matrix_col,sizeof(int *));
			   hipMemcpyToSymbol(HIP_SYMBOL(dg_csr_index_data),&d_csr_index_data,sizeof(int *));
			   hipMemcpyToSymbol(HIP_SYMBOL(dg_global_matrix_ptr),&d_global_matrix_ptr,sizeof(int *));
cout<<"Elapsed time in hipMemcpy "<<clock.elapsed()<<endl;	
//assert(result==hipSuccess);
//cout<<d_element_color_data[0]<<endl;
//dim3 threadperblock(2,3);
/////////////////////////////////////////////******************Kernel launch for Assembly and boundary condition.
//dim3 blocks(1,1);
 k=0;
int noofblocks=ceil((float)colors_size[0]/64);
cout<<"No of blocks :"<<noofblocks<<endl;
dim3 blocks(noofblocks);
dim3 threadperblock(64);


//clock.start();
for(int i=0;i<8;i++)
{

Assembly<<<blocks,threadperblock>>>(d_element_color_data+k,colors_size[i],d_material_data,k,d_element_matrix,d_rho,penal);
k+=colors_size[i];
//cout<<k<<endl;
//hipDeviceSynchronize();
//hipError_t error=hipDeviceSynchronize();
//if(error!=hipSuccess)
//cerr<<"error::"<<hipGetErrorString(error)<<endl;
}

//cout<<"elapsed time in kernel :"<<clock.elapsed()<<endl;

noofblocks=((z_nelement+1)*(y_nelement+1)*3)/257+1;
threadperblock.x=256;
cout<<"No of boundary nodes "<<(z_nelement+1)*(y_nelement+1)*3<<endl;
cout<<"No of blocks boundary "<<noofblocks<<endl;
blocks.x=noofblocks;
boundary_condtn<<<noofblocks,threadperblock>>>(d_boundary_nodes,(z_nelement+1)*(y_nelement+1)*3,n_node,end_load,y_nelement+1,d_global_vector);

//hipDeviceSynchronize();
//hipDeviceSynchronize();
//cout<<"GPU memory usage :"<<GetGPUmeminfo()<<endl;
			//hipMemcpy(global_matrix_data,d_global_matrix_data,sizeof(float)*nnz,hipMemcpyDeviceToHost);
//			hipMemcpy(global_matrix_col,d_global_matrix_col,sizeof(int)*nnz,hipMemcpyDeviceToHost);
//***************************************************************************************************************

		hipFree(d_element_color_data);
		hipFree(d_nodal_data);
		hipFree(d_connect_matrix);
		hipFree(d_material_data);
		//hipFree(d_global_matrix_data);
		//hipFree(d_global_matrix_col);
		//hipFree(d_global_matrix_ptr);
		hipFree(d_csr_index_data);
		hipFree(d_boundary_nodes);
		//hipFree(d_global_vector);
//print(global_matrix_data,576,24);
//element_stiffness(4);
//print(elemental_data,576,24);

	

/*	for(int i=0;i<n_node/2;i++)
		{for(int j=0;j<(global_rowptr[i+1]-global_rowptr[i]);j++)
		cout<<global_matrix_data[global_rowptr[i]+j]<<"\t";
		cout<<endl;
		}
*/	

//print(sol,n_node,1);
//******************************************************************************************CUSP
	//thrust::device_ptr<float> wrapped_b_vect(d_global_vector);
	typedef typename cusp::array1d_view<thrust::device_ptr<int> > DeviceIndexArrayView;
    	typedef typename cusp::array1d_view<thrust::device_ptr<float> > DeviceValueArrayView;
	
		thrust::device_ptr<float> wrapped_b_vect(d_global_vector);	
		thrust::device_ptr<float> wrapped_d_global_matrix_data(d_global_matrix_data);	
		thrust::device_ptr<int>   wrapped_d_global_matrix_col(d_global_matrix_col);
		thrust::device_ptr<int>	  wrapped_d_global_matrix_ptr(d_global_matrix_ptr);
		thrust::device_ptr<float> wrapped_d_sol(d_sol);
	DeviceIndexArrayView row_offsets (wrapped_d_global_matrix_ptr,wrapped_d_global_matrix_ptr+n_node+1);
	DeviceIndexArrayView col_indices (wrapped_d_global_matrix_col,wrapped_d_global_matrix_col+nnz);
	DeviceValueArrayView values (wrapped_d_global_matrix_data,wrapped_d_global_matrix_data+nnz);
	DeviceValueArrayView sol_d  (wrapped_d_sol,wrapped_d_sol+n_node);
	//HostValueArrayView x (sol,sol+n_node);
	//DeviceValueArrayView b_vect (d_global_vector,d_global_vector+n_node);
	DeviceValueArrayView b_vect(wrapped_b_vect,wrapped_b_vect+n_node);
	typedef cusp::csr_matrix_view<DeviceIndexArrayView,DeviceIndexArrayView,DeviceValueArrayView> DeviceView;
	DeviceView A(n_node,n_node,nnz,row_offsets,col_indices,values);	

	//cusp::array1d<float,cusp::device_memory>x(n_node,0);
//hipDeviceSynchronize();	
	//cusp::print(b_vect);
	//cusp::print(A.values);
//	cusp::default_monitor<float> monitor(b_vect, 10000, 1e-5);
	cusp::monitor<double> monitor(b_vect,2000,1e-5);
	cusp::krylov::cg(A,sol_d,b_vect,monitor);
//	print(sol_d);
	//cout<<x(n_node);
	float *sol=new float[n_node];
	hipMemcpy(sol, d_sol,sizeof(float)*n_node,hipMemcpyDeviceToHost);
	for(int i=0;i<n_node;i++) u[i]=sol[i];
	delete [] sol;
//hipDeviceSynchronize();
//	cusp::print(b_vect);
//	print(sol_d);



	hipFree(d_global_vector);
	hipFree(d_global_matrix_data);
	hipFree(d_global_matrix_col);
	hipFree(d_global_matrix_ptr);
	hipFree(d_sol);


//*********************************************************************************************************************
//cout<<"last node "<<2*(h_nelement*v_nelement+1)+1<< "y displacement "<<sol[2*(h_nelement*v_nelement+1)+1]<<"\n";
/*	ostringstream outfile("");
	outfile<<"file_"<<d_length<<".dat";	
	string filename=outfile.str();
	cout<<filename<<endl;
	ofstream outf(filename.c_str(),ios::app);
	//outf<<h_nelement*v_nelement<<"\t"<<sol[2*(h_nelement*v_nelement+1)+1]<<"\n";
	int testnode=h_nelement*((v_nelement/2)+1)+(v_nelement/2);
	outf<<h_nelement*v_nelement<<"\t"<<sol[2*testnode+1]<<"\n";
*/	//cout<<testnode<<endl;
//delete []re_nodal_data;
//delete []re_connect_matrix;
//delete []boundary_nodes;
//delete []csr_index_data;
//delete []re_csr_index_data;
//delete []global_matrix_data;
//delete []global_matrix_col;
//delete []global_rowptr;
//delete []global_vector;
//delete []material_data;
//delete []elemental_data;
//delete []elemental_vector;
//delete []element_color;
//delete []element_color_data;
//delete []Nodes_noofelements;
//delete []Nodesharingelements;
//delete []Nodesharingelements_ptr;
//delete []nodal_data;
//delete []connect_matrix;
//delete []sol;
//return 0;
}
template<typename T>
void print(T *data,int size,int rowlength)
{
for(int i=0;i<size/rowlength;i++)
{
	for(int j=0;j<rowlength;j++)
		cout<<data[i*rowlength+j]<<"\t";
	cout<<"\n";
}
}

void print(float** data,int size)
{
for(int i=0;i<size;i++)
{
        for(int j=0;j<size;j++)
                cout<<data[i][j]<<"\t";
        cout<<"\n";
}


}



void print(int h_nelement,int v_nelement,int z_nelement)
{
	for(int i=0;i<h_nelement*v_nelement*z_nelement;i=i+1)
	{
	cout<<"values of i"<<i<<endl;
	for(int j=0;j<8;j++)
	cout<<connect_matrix[i*8+j];
	cout<<"\n";
	}
	for(int i=0;i<3*(h_nelement+1)*(v_nelement+1)*(z_nelement+1);i=i+3)
	cout<<nodal_data[i]<<"\t"<<nodal_data[i+1]<<"\t"<<nodal_data[i+2]<<"\n";

}

void Calc_global_rowptr()
{
 	global_rowptr=new int [n_node+1];
		global_rowptr[0]=0;
                global_rowptr[1]=24;
                global_rowptr[2]=48;
                global_rowptr[n_node]=nnz;
                for(int j=1;j<=n_node/3;j++)
                {
                int k=0;
                if(Nodes_noofelements[j-1]==1)
                {k=24;}
                else if(Nodes_noofelements[j-1]==2)
                k=36;
                else if(Nodes_noofelements[j-1]==4)
                k=54;
                else if(Nodes_noofelements[j-1]==8)
                k=81;
                //global_rowptr[j]= Nodes_noofelement[j-1]*
                global_rowptr[3*j-2]=global_rowptr[3*j-3]+k;
                global_rowptr[3*j-1]=global_rowptr[3*j-2]+k;
                global_rowptr[3*j]=global_rowptr[3*j-1]+k;
                //global_rowptr[3*j+1]=global_rowptr[3*j]+k;
                //global_rowptr[3*j+2]=global_rowptr[3*j+1]+k;
                }




}

float GetGPUmeminfo()
{

	size_t free_byte ;
	size_t total_byte ;

	hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte );

	if ( hipSuccess != cuda_status ){
	printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
	exit(1);
	}

float free_db = (float)free_byte ;
float total_db = (float)total_byte ;
float used_db = total_db - free_db ;

//	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	return used_db/1024/1024;
}



