#include "hip/hip_runtime.h"
#include"print_fn.h"
#include<stdio.h>
#include"header.h"
//extern __device__ int *color_data;
//template<typename T>
//__device__ void print(T *array,int nrows,int size);
template<typename T>
__device__ void printc(T *array,int nrows,int size)
{
	for(int i=1;i<=size;i++)
	{
		printf("%f ",array[i-1]);
		if((i)%nrows==0)
		printf("\n");
	}

}


__global__ void Assembly(int *element_color_data,int mxelement,float *d_material_data,int connect_index, float *d_element_matrix, double* d_rho, double penal )
{
//	if(threadIdx.x==0) printf("max element %d\n",mxelement);
	//int local_elementno=threadIdx.x;
	int local_elementno=(gridDim.x*blockIdx.y*blockDim.x*blockDim.y)+(blockIdx.x*blockDim.x*blockDim.y)+ blockDim.x*threadIdx.y+threadIdx.x;
	int local_threadid=blockDim.x*threadIdx.y+threadIdx.x;
//	printf("local element %d\n",local_elementno);
	if(local_elementno < mxelement)
	{
		int global_elementno=element_color_data[local_elementno]+1;
		int node_connect[8];	//for 8 noded brick element 
		//int csr_index[64];
//	printf("global no %d\n",global_elementno);	
		for(int i=0;i<8;i++)
		{node_connect[i]=dg_connect_matrix[connect_index*8+i*mxelement+local_elementno];
		//printf("%d\n",node_connect[i]);
		}
//printf("thread %d ,index %d\n",local_elementno,dg_connect_matrix[connect_index*8+0*mxelement+local_elementno]);
		//if(global_elementno==15)
		//printc(node_connect,1,8);
	 float elemental_data[576];
	for(int i=0;i<576;i++) elemental_data[i]=0;
//		for(int i=local_threadid;i<576;i=i+blockDim.x*blockDim.y) elemental_data[i]=d_element_matrix[i];
		//if(local_threadid==0) 
		//for(int i=0;i<576;i++) elemental_data[i]=d_element_matrix[i];
//__syncthreads();
	element_stiffness(local_elementno,d_material_data,elemental_data,node_connect,connect_index,mxelement);
//__syncthreads();
		//if(global_elementno==33767)
		//printc(elemental_data,24,576);
		//#pragma unroll	
		for(int i=0;i<8;i++)
		{
			
			for(int j=0;j<24;j++)
			{
			dg_global_matrix_data[dg_global_matrix_ptr[3*node_connect[i]]+3*dg_csr_index_data[(connect_index)*64+i*8*mxelement+(j/3)*mxelement+local_elementno]+j%3]  += elemental_data[3*i*24+j]*powf(d_rho[global_elementno-1],penal);
			dg_global_matrix_data[dg_global_matrix_ptr[3*node_connect[i]+1]+3*dg_csr_index_data[(connect_index)*64+i*8*mxelement+(j/3)*mxelement+local_elementno]+j%3]+=elemental_data[(3*i+1)*24+j]*powf(d_rho[global_elementno-1],penal); 
			dg_global_matrix_data[dg_global_matrix_ptr[3*node_connect[i]+2]+3*dg_csr_index_data[(connect_index)*64+i*8*mxelement+(j/3)*mxelement+local_elementno]+j%3]+=elemental_data[(3*i+2)*24+j]*powf(d_rho[global_elementno-1],penal); 
			//printf("%d ",3*dg_csr_index_data[(global_elementno-1)*64+i*8+j/3]+j%3);
			dg_global_matrix_col[dg_global_matrix_ptr[3*node_connect[i]]+3*dg_csr_index_data[(connect_index)*64+i*8*mxelement+(j/3)*mxelement+local_elementno]+j%3]=3*node_connect[j/3]+j%3;
			dg_global_matrix_col[dg_global_matrix_ptr[3*node_connect[i]+1]+3*dg_csr_index_data[(connect_index)*64+i*8*mxelement+(j/3)*mxelement+local_elementno]+j%3]=3*node_connect[j/3]+j%3;
			dg_global_matrix_col[dg_global_matrix_ptr[3*node_connect[i]+2]+3*dg_csr_index_data[(connect_index)*64+i*8*mxelement+(j/3)*mxelement+local_elementno]+j%3]=3*node_connect[j/3]+j%3;
	//	__syncthreads();
			}
		//__syncthreads();
//printf("\n");
		}

//__syncthreads();

//printf("hello\n");
//printf("%d\n",element_color_data[threadIdx.x]);
//printf("color data %d\n",color_data[threadIdx.x]);
//printf("%p\n",color_data);
//printf("%p\n",element_color_data);
//if(threadIdx.x==0)
//printf("global element no %d\n",global_elementno);
//printf("value %d\n",2/3);
//if(global_elementno==3)
//printc(dg_global_matrix_col,24,576);

//printf("%f %f\n",elemental_data[0],elemental_data[1]);
	}
//printf("hello\n");
//__syncthreads();






}
