#include "hip/hip_runtime.h"
#include<stdio.h>
#include<cassert>
#include<cmath>
#include"header.h"
using namespace std;
//__device__ double jacobian[9];
//__device__ double matrix_b[144];  ///(6x24 matrix contains derivative of shape function in physical coordinates)
///elemental computation uses isoparametric shape function 

__device__ void element_stiffness(int elementno,float *d_material_data,float *elemental_data,int *node_connect,int node_index,int mxelement)
{
	float jacobian[9];
	float matrix_b[144];
	double *gausspoints=new double [nint];
	double *weights=new double [nint];
	getGausspoints(gausspoints,weights,nint);
//printf("gausspoints : %f %f\n",gausspoints[0],gausspoints[1]);	
	float d_shapefn[24];
//	printf("elementno :%d\n",elementno);	
for(int m=0;m<nint;m++)
{

for(int n=0;n<nint;n++)
{
	
for(int p=0;p<nint;p++)
{
//	cout<<"LOOP"<<"\n";
	//cout<<"Points "<<gausspoints[m]<<"\t"<<gausspoints[n]<<"\t"<<gausspoints[p]<<"\n";
	for(int i=0;i<144;i++) {matrix_b[i]=0;}// initialising

	float determinant=0;
///////////////////////////keeping values of derivative of shape function in an array
	for(int i=0;i<24;i++)
{	if(i<8)
	d_shapefn[i]=d_phi(i+1,gausspoints[m],gausspoints[n],gausspoints[p],'z');
	else if(i>=8 && i<16)
	d_shapefn[i]=d_phi(i-7,gausspoints[m],gausspoints[n],gausspoints[p],'e');
	else 
	d_shapefn[i]=d_phi(i-15,gausspoints[m],gausspoints[n],gausspoints[p],'j');
}

//__syncthreads();
/*
if(elementno==1)
{	for(int i=0;i<3;i++)
	{
	for(int j=0;j<8;j++)
	{printf("%f  ",d_shapefn[i*8+j]);}
	printf("hello\n");
	}}*/
//__syncthreads();
////////////////calculating the jacobian value
	jacobian[0]=jacobian[1]=jacobian[2]=jacobian[3]=0;		//initialising 
	jacobian[4]=jacobian[5]=jacobian[6]=jacobian[7]=jacobian[8]=0;
//__syncthreads();	
	for(int j=0;j<8;j++)
		{
		//cout<<"node :"<<nodal_data[3*(connect_matrix[(elementno-1)*8+j])]<<nodal_data[3*(connect_matrix[(elementno-1)*8+j])+2];
		float x,y,z;
		x=dg_nodal_data[node_index*24+j*mxelement*3+elementno];
		y=dg_nodal_data[node_index*24+j*mxelement*3+elementno+mxelement];
		z=dg_nodal_data[node_index*24+j*mxelement*3+elementno+2*mxelement];
		//x=dg_nodal_data[3*node_connect[(j)]];
		//y=dg_nodal_data[3*(node_connect[j])+1];
		//z=dg_nodal_data[3*(node_connect[j])+2];
		jacobian[0]=jacobian[0]+d_shapefn[j]*x;
		jacobian[3]=jacobian[3]+d_shapefn[j+8]*x;
		jacobian[6]=jacobian[6]+d_shapefn[j+16]*x;
		jacobian[1]=jacobian[1]+d_shapefn[j]*y;
		jacobian[4]=jacobian[4]+d_shapefn[j+8]*y;
		jacobian[7]=jacobian[7]+d_shapefn[j+16]*y;
		jacobian[2]=jacobian[2]+d_shapefn[j]*z;
		jacobian[5]=jacobian[5]+d_shapefn[j+8]*z;
		jacobian[8]=jacobian[8]+d_shapefn[j+16]*z;
		}
//__syncthreads();	
////////////////////////////***********Determinant
	determinant=jacobian[0]*(jacobian[4]*jacobian[8]-jacobian[5]*jacobian[7])-jacobian[1]*(jacobian[3]*jacobian[8]-jacobian[5]*jacobian[6])+jacobian[2]*(jacobian[3]*jacobian[7]-jacobian[4]*jacobian[6]);

//	printf("determinant :%f\n",determinant);

/*	cout<<"Jacobian "<<endl<<endl;
	for(int i=0;i<3;i++)
        {
        for(int j=0;j<3;j++)
	cout<<jacobian[i*3+j]<<" ";
	cout<<endl;
	}
*/



////////**************************************inverse of jacobian
{
	float k[9];
	k[0]=(jacobian[4]*jacobian[8]-jacobian[5]*jacobian[7]);
	k[1]=-1*(jacobian[3]*jacobian[8]-jacobian[5]*jacobian[6]);	
	k[2]=(jacobian[3]*jacobian[7]-jacobian[4]*jacobian[6]);
	k[3]=-1*(jacobian[1]*jacobian[8]-jacobian[2]*jacobian[7]);
	k[4]=(jacobian[0]*jacobian[8]-jacobian[2]*jacobian[6]);
	k[5]=-1*(jacobian[0]*jacobian[7]-jacobian[1]*jacobian[6]);
	k[6]=(jacobian[1]*jacobian[5]-jacobian[2]*jacobian[4]);
	k[7]=-1*(jacobian[0]*jacobian[5]-jacobian[2]*jacobian[3]);
	k[8]=(jacobian[0]*jacobian[4]-jacobian[1]*jacobian[3]);
	jacobian[0]=k[0]/determinant;
	jacobian[3]=k[1]/determinant;
	jacobian[6]=k[2]/determinant;
	jacobian[1]=k[3]/determinant;
	jacobian[4]=k[4]/determinant;
	jacobian[7]=k[5]/determinant;
	jacobian[2]=k[6]/determinant;
	jacobian[5]=k[7]/determinant;
	jacobian[8]=k[8]/determinant;

}
/*	cout<<"Inverse of jacobian "<<endl<<endl;
	for(int i=0;i<3;i++)
        {
        for(int j=0;j<3;j++)
	cout<<jacobian[i*3+j]<<" ";
	cout<<endl;
	}

*/

////////////////////////////////////////////////calculating matrix_b
for(int i=0;i<8;i++)
{

	matrix_b[i*3]=jacobian[0]*d_shapefn[i]+jacobian[1]*d_shapefn[i+8]+jacobian[2]*d_shapefn[i+16];
	//cout<<jacobian[3]<<" "<<jacobian[4]<<" "<<jacobian[5]<<""<<endl;
	//cout<<d_shapefn[i]<<" "<<d_shapefn[i+8]<<" "<<d_shapefn[i+16]<<endl;
	matrix_b[i*3+25]=jacobian[3]*d_shapefn[i]+jacobian[4]*d_shapefn[i+8]+jacobian[5]*d_shapefn[i+16];
	matrix_b[i*3+50]=jacobian[6]*d_shapefn[i]+jacobian[7]*d_shapefn[i+8]+jacobian[8]*d_shapefn[i+16];
	matrix_b[i*3+73]=jacobian[6]*d_shapefn[i]+jacobian[7]*d_shapefn[i+8]+jacobian[8]*d_shapefn[i+16];
	matrix_b[i*3+74]=jacobian[3]*d_shapefn[i]+jacobian[4]*d_shapefn[i+8]+jacobian[5]*d_shapefn[i+16];
	matrix_b[i*3+96]=jacobian[6]*d_shapefn[i]+jacobian[7]*d_shapefn[i+8]+jacobian[8]*d_shapefn[i+16];
	matrix_b[i*3+98]=jacobian[0]*d_shapefn[i]+jacobian[1]*d_shapefn[i+8]+jacobian[2]*d_shapefn[i+16];
	matrix_b[i*3+120]=jacobian[3]*d_shapefn[i]+jacobian[4]*d_shapefn[i+8]+jacobian[5]*d_shapefn[i+16];
	matrix_b[i*3+121]=jacobian[0]*d_shapefn[i]+jacobian[1]*d_shapefn[i+8]+jacobian[2]*d_shapefn[i+16];

}
/*	cout<<"\n\n";
	for(int i=0;i<6;i++)
		{for(int j=0;j<24;j++)
		cout<<matrix_b[i*24+j]<<"  ";
		cout<<"\n";}
*/
/////////////////////////////////multiplication B'CB(~~~~~~~another way is possible(suitable for GPU) in which each non zero element is calculated individually without using matrix_b)
for(int i=0;i<24;i++)
{
	float a=0,b=0,c=0,d=0,e=0,f=0;
	a=(matrix_b[i]*d_material_data[0]+matrix_b[i+24]*d_material_data[6]+matrix_b[i+48]*d_material_data[12]+matrix_b[i+72]*d_material_data[18]+matrix_b[i+96]*d_material_data[24]+matrix_b[i+120]*d_material_data[30]);
	b=(matrix_b[i]*d_material_data[1]+matrix_b[i+24]*d_material_data[7]+matrix_b[i+48]*d_material_data[13]+matrix_b[i+72]*d_material_data[19]+matrix_b[i+96]*d_material_data[25]+matrix_b[i+120]*d_material_data[31]);
	c=(matrix_b[i]*d_material_data[2]+matrix_b[i+24]*d_material_data[8]+matrix_b[i+48]*d_material_data[14]+matrix_b[i+72]*d_material_data[20]+matrix_b[i+96]*d_material_data[26]+matrix_b[i+120]*d_material_data[32]);
	d=(matrix_b[i]*d_material_data[3]+matrix_b[i+24]*d_material_data[9]+matrix_b[i+48]*d_material_data[15]+matrix_b[i+72]*d_material_data[21]+matrix_b[i+96]*d_material_data[27]+matrix_b[i+120]*d_material_data[33]);
	e=(matrix_b[i]*d_material_data[4]+matrix_b[i+24]*d_material_data[10]+matrix_b[i+48]*d_material_data[16]+matrix_b[i+72]*d_material_data[22]+matrix_b[i+96]*d_material_data[28]+matrix_b[i+120]*d_material_data[34]);
	f=(matrix_b[i]*d_material_data[5]+matrix_b[i+24]*d_material_data[11]+matrix_b[i+48]*d_material_data[17]+matrix_b[i+72]*d_material_data[23]+matrix_b[i+96]*d_material_data[29]+matrix_b[i+120]*d_material_data[35]);

//	cout<<"a "<<a<<" b "<<b<<" c "<<c<<" d "<<d<<" e "<<e<<" f "<<f<<"\n";
	for(int j=0;j<24;j++)
	{ 
		elemental_data[i*24+j]=elemental_data[i*24+j]+(a*matrix_b[j]+b*matrix_b[j+24]+c*matrix_b[j+48]+d*matrix_b[j+72]+e*matrix_b[j+96]+f*matrix_b[j+120])*determinant*weights[m]*weights[n]*weights[p];
	}

}
//__syncthreads();
/*	cout<<"\n\n";
	for(int i=0;i<24;i++)
		{for(int j=0;j<24;j++)
		cout<<elemental_data[i*24+j]<<"  ";
		cout<<"\n";}
*/
	
	//******************calculating source vector************
/*	for(int i=0;i<4;i++)
	{
		elemental_vector[2*i]+=phi(i+1,gausspoints[m],gausspoints[n],gausspoints[p])*determinant*weights[m]*weights[n]*0;
		elemental_vector[2*i+1]+=phi(i+1,gausspoints[m],gausspoints[n],gausspoints[p])*determinant*weights[m]*weights[n]*0;
	}*/
}
//__syncthreads();
}
//__syncthreads();
}


delete []gausspoints;
delete []weights;
}



/////////////////////////////////////////gives derivative of linear interpolation

/*double d_phi(int i,double point,char type)
{
assert(i>=1 && i<=4);
if(type=='z')
	{
	if(i==1 ||i==4)
		return (-1+pow(-1,i+1)*point)/4;
	return (1+point*pow(-1,i+1))/4;

	}
else
	{
	if(i==1 ||i==2)
		return (-1+point*pow(-1,i+1))/4;
	return (1+point*pow(-1,i+1))/4;	

	}

}*/

__device__ double d_phi(int i,double zeta,double eta,double zi,char type)
	{
	double k=0;
	double ret;
	assert(i>=1 && i<=8);
	switch(type)
	{
		case 'z':
			k=phi(i,zeta,eta,zi);
			if(i==1 || i==4 || i==5 || i==8)	
			ret=-1*k/(1-zeta);
			else
			ret=k/(1+zeta);
			break;
		case 'e':
			k=phi(i,zeta,eta,zi);
			if(i==1 || i==2 || i==5 || i==6)
			ret=-1*k/(1-eta);
			else
			ret=k/(1+eta);
			break;
		case 'j':
			k=phi(i,zeta,eta,zi);
			if(i==1 || i==2 || i==3 || i==4)
			ret=-1*k/(1-zi);
			else
			ret=k/(1+zi);
			break;
  		default :
			ret=0;//MA 	
			break;
	}
	return ret;



	}


///////////////////////////////////gives linear interpolation function
__device__ double phi(int i,double zeta,double eta,double zi)
{
assert(i>=1 && i<=8);
double ret=0;
switch(i)
{
	case 1:
		ret=0.125*(1-zeta)*(1-eta)*(1-zi);
		break;
	case 2:
		ret=0.125*(1+zeta)*(1-eta)*(1-zi);
		break;
	case 3:
		ret= 0.125*(1+zeta)*(1+eta)*(1-zi);
		break;
	case 4:
		ret= 0.125*(1-zeta)*(1+eta)*(1-zi);
		break;
	case 5:
		ret= 0.125*(1-zeta)*(1-eta)*(1+zi);
		break;
	case 6: 
		ret= 0.125*(1+zeta)*(1-eta)*(1+zi);
		break;
	case 7:
		ret= 0.125*(1+zeta)*(1+eta)*(1+zi);
		break;
	case 8:
		ret= 0.125*(1-zeta)*(1+eta)*(1+zi);
		break;
	default:
			
		break;

}
return ret;
}
