#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"header_cpp.h"
#include"clock.h"
#include<fstream>
#include<iostream>
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((a)>(b)?(a):(b))
void FE(int x_nelement, int y_nelement, int z_nelement, int loop, double * rho, double penal);
void OCupdate(int nx,int ny,int nz,double *rho,double volfrac,double MinDens,double *d_comp);
void check(int nx,int ny,int nz,double rmin,double *rho, double *d_comp);
      
        float *nodal_data;
        int *connect_matrix;
        int *element_color;
        double *elemental_data;  //stores elemental stiffness matrix
        float *elemental_vector;  //stores elemental source vector
        float *material_data;   //stores material properties
        //float **global_matrix;        //stores global stiffness matrix
        int *Nodes_noofelements;//keeps the no of elements containing a particular node
        int *Nodesharingelements;// keeps the list of element number sharing a node
        int *Nodesharingelements_ptr;
        float *global_matrix_data;
        int *global_matrix_col;
        int *global_rowptr;
        float * global_vector;  //stores global source vector
        //float * sol;
        int n_node;
        int nnz;
        int nelement;
	double *u;
//      int *boundary_nodes;

int main()
{
StopWatch_CPU cpu_clock;
StopWatch_GPU feTime;
StopWatch_GPU OCTime;
StopWatch_GPU CheckTime;
int nx = 200;
int ny = 100;
int nz = 100;
int n_node= (nx+1)*(ny+1)*(nz+1);
int n_elem= nx*ny*nz;
	//elemental_data=		new float[576];////////for linear shape function  on rectangular element having 3 degree of freedom per node..24*24
	//elemental_vector=	new float[24];
			
/*	sol= new float[3*n_node];
	for(int i=0;i<3*n_node;i++)
	{ 
	sol[i]=0;
	}
*/

	elemental_data     =    new double[576];
	nodal_data = new float[3*n_node];
        connect_matrix = new int[8*n_elem];
        element_color = new int[n_elem];
	data_read(n_node, n_elem);


double volfrac = 0.3;
double penal = 3;
double rmin = 3.0;
double MinDens = 0.1;

u = (double *)malloc(3*n_node * sizeof(double));
for(int i = 0; i < 3*n_node; i++)
{
	u[i] = 1.0;
}
double ue[24];
double Ke[24][24];
double uKe[24];
double uKeu = 0.0;

for(int i = 0; i < 24; i++)
{
	ue[i] = 0.0;
	uKe[i] = 0.0;
	for(int j = 0; j < 24; j++)
	{
		Ke[i][j] = 0.0;
		
	}
}
double comp = 0.0; //Compliance
double *d_comp = (double *)malloc(nx*ny*nz*sizeof(double));
for(int i = 0; i < nx*ny*nz; i++)
{
	d_comp[i] = 0.0;
}

int *connectMatrix = (int *)malloc(nx*ny*nz*8*sizeof(int));
for(int i = 0; i < nx*ny*nz*8; i++)
{
	connectMatrix[i] = connect_matrix[i];
}

double *rho =(double *) malloc(nx*ny*nz*sizeof(double));
double *rho_old = (double *)malloc(nx*ny*nz*sizeof(double));

for(int i = 0; i < nx*ny*nz; i++)
{
	rho[i] = volfrac;
	rho_old[i] = volfrac;
}

int loop = 0;
double change = 1;

double checkMax = 0.0;
double checkSum = 0.0;
//Main loop

cpu_clock.start();
while( change > 0.01 && loop <5)
{
	//viz(u,rho,connectMatrix,loop);
	loop++;
	printf("\n Loop no :%d\nCompliance = %lf\n",loop,comp);
	for(int i = 0; i < nx*ny*nz ; i++)
	{
		rho_old[i] = rho[i];
	}
	
	///////////// u = FEA();
	feTime.start();
	FE(nx,ny,nz,loop,rho,penal);
	feTime.stop();
	//for(int i=0;i<3*n_node;i++) printf("%lf \n",u[i]);
	if(loop==1)
	{
	for(int i=0;i<24;i++)
	for(int j=0;j<24;j++){
      	Ke[i][j]=elemental_data[i*24+j];
		}
	}
	///////////// Ke = FEA();
	
	comp = 0.0;

	for(int i = 0; i < nx * ny * nz; i++)
	{
		for(int ii = 0; ii < 8; ii++)
		{
			ue[ii * 3] =  u[connectMatrix[i * 8 + ii] * 3];
			ue[ii * 3 + 1] = u[connectMatrix[i * 8 + ii] * 3 + 1];
			ue[ii * 3 + 2] = u[connectMatrix[i * 8 + ii] * 3 + 2];
			
		}
//		if(i == 167){for(int j = 0; j < 24; j++) printf("\nUe = %lf", ue[j]);}
		for(int j = 0; j <24; j++)
		{
			uKe[j] = 0.0;
		}
		for(int j = 0; j < 24; j++)
		{
			for(int k = 0; k < 24; k++)
			{
				uKe[j] += ue[k] * Ke[k][j]; 
			}
		}
	/*	if(i == 167)
		{
			printf("\n");	
			for(int l = 0; l < 24; l++)
			{
				for(int m = 0; m < 24; m++)
				{
					printf("%lf ",Ke[l][m]);
				}
				printf("\n");
			}
		}
		*/
 //		if(i == 167){for(int j = 0; j < 24; j++) printf("\nUKe = %lf", uKe[j]);}
		uKeu = 0.0;
		for(int j = 0; j < 24; j++)
		{
			uKeu += uKe[j] * ue[j];
		}
//		if(i == 167){for(int j = 0; j < 24; j++) printf("\nUe = %lf", ue[j]);}
//		{printf("\ni = %d UKeu = %lf",i, uKeu);}
		comp = comp + pow(rho[i], penal) * uKeu;
		d_comp[i] = - penal * pow(rho[i], (penal - 1)) * uKeu;
	}
	
	printf("\n Second Compliance = %lf\n",comp);	
	//Sensititivity filter
//	for(int i=0;i<nx * ny * nz;i++) printf("\n d_comp = %lf\n",d_comp[i]);
	/////////////////////////////////////////////////////////////////////////////////////////////////////




	CheckTime.start();
	check(nx,ny,nz,rmin,rho,d_comp);
	CheckTime.stop();
//	for(int i=0;i<nx * ny * nz;i++) printf("\n updated_d_comp = %lf\n",d_comp[i]);


	//////////////////////////////////////////////////////////////////////////////////////////////////////
	//Design update
	OCTime.start();
	OCupdate(nx,ny,nz,rho,volfrac,MinDens,d_comp);
	OCTime.stop();
//	for(int i=0;i<nx * ny * nz;i++)	printf("\n Rho = %lf\n",rho[i]);	
	
	for(int i = 0; i < nx * ny * nz; i++)
	{
		if(abs(rho[i] - rho_old[i]) > checkMax)
		{
			checkMax = abs(rho[i] - rho_old[i]);
		}
	}
	
	change = checkMax;


	printf("\n				FE :%lf\n                              Check :%lf\n                              OC :%lf",feTime.elapsed(),CheckTime.elapsed(),OCTime.elapsed());		

	
//	change = 0;
}
	printf("Time taken in 3 iteration: %lf \n",cpu_clock.elapsed());

	std::ofstream myfile;
        myfile.open("rho2.txt");
        if(myfile.is_open())
        {
                for(int i=0;i< nx * ny * nz;i++){
                myfile<<rho[i]<<"\n";
                }
        myfile.close();
        }
	
	myfile.open("disp2.txt");
        if(myfile.is_open())
        {
                for(int i=0;i< 3*n_node ;i++){
                myfile<<u[i]<<"\n";
                }
        myfile.close();
        }




return 0;
}

void check(int nx,int ny,int nz,double rmin,double *rho, double *d_comp)
{
   double *New_d_comp = (double*)malloc(sizeof(double)*nx*ny*nz);
  
  	for(int i=0; i<nx*ny*nz; i++)
  	{  New_d_comp[i] = 0.0;     }
  	
  	for(int k=0; k<nz; k++)
  	{
  	   for(int j=0; j<ny; j++)
  	   {
  	      for(int i=0; i<nx; i++)
  	      {
  	      	   double sum = 0.0;
  	       	   // Global no. of this element
  	      	   int id1 = i+(j*nx)+(k*nx*ny);
  	      	   
  	      	   for(int ek = max(k-(int)rmin,0); ek<= min(k+(int)rmin,nz); ek++)
  	      	   {
  	      	      for(int ej = max(j-(int)rmin,0); ej<= min(j+(int)rmin,ny); ej++)
  	      	      {
  	      	          for(int ei = max(i-(int)rmin,0); ei<= min(i+(int)rmin,nx); ei++)
  	      	          {	
  	      	              // Global no. of this element
  	       	              int id2 = ei+(ej*nx)+(ek*nx*ny);
  	      	              
  	      	              double fac = rmin-sqrt((k-ek)*(k-ek)+(j-ej)*(j-ej)+(i-ei)*(i-ei));
              		      sum += max(0.0, fac);
//              		      printf("sum %lf\n",sum); 
              		      New_d_comp[id1] += d_comp[id2]*max(0.0, fac)*rho[id2];
              		      
  	      	          }
  	      	      }
  	      	   }
  	         
  	         New_d_comp[id1] =  New_d_comp[id1]/(rho[id1]*sum);
  	      }
  	   }
  	}
  	
  	for(int i=0; i<nx*ny*nz; i++)
  	{  
  	   d_comp[i] = New_d_comp[i];     
    	}
  printf("Check working");
}

void OCupdate(int nx,int ny,int nz,double *rho,double volfrac,double MinDens,double *d_comp)
{
  double l1 = 0.0;
  double l2 = 1e5;
  int counter = 0;	// Bi-section iteration counter
  double move = 0.2;
  
  double *New_rho = (double*)malloc(sizeof(double)*nx*ny*nz);
  
  	for(int i=0; i<nx*ny*nz; i++)
  	{  New_rho[i] = 0.0;     }
    	double derivative;
	while(l2-l1 > 1e-4 && counter < 1e3)
	{
		   counter++;
		   
	    	  // Bi-section 
	    	  double lmid = 0.5*(l2+l1);
	    	  
	    	  //loop over all elements
		  for(int i=0; i<nx*ny*nz; i++)
	    	  {
	    	       //the gradient must be less than zero everywhere, so
          	       //everything bigger than 0.0 will be truncated
          	       derivative = max(-1.0*d_comp[i], 0.0);
          	       
          	       // Updated d_comp
          	       New_rho[i] = max(MinDens, max(rho[i]-move, min(1.0, min(rho[i]+move, rho[i]*sqrt(derivative/lmid)))));
          	  }
	    	   
	    	   // Summing the gradients
	    	   double sum = 0.0;
	    	   for(int i=0; i<nx*ny*nz; i++)
	    	   {
	    	   	sum += New_rho[i];
	    	   }
	    	 	    	 
	    	 if(sum - volfrac*nx*ny*nz > 0)
      		   {	l1 = lmid;	}
    		 else
      		   {	l2 = lmid;	}
		  
		 // Updating
		 
	    	 
   //free(New_d_comp);
	}
	for(int i=0; i<nx*ny*nz; i++)
                 {
                    rho[i] = New_rho[i];
                 }
	printf("OC working");
}
